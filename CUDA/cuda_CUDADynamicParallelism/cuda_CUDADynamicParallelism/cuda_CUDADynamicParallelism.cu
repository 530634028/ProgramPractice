#include "hip/hip_runtime.h"

/****************************************
*
*  date: 2018-5-23
*  a   : zhonghy
*
*
*
*****************************************/

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>


//simple Hello World program incorporating dynamic parallelism
__global__ void childKernel()
{
	printf("Hello ");
}

__global__ void parentKernel()
{
	//launch child
	childKernel<<<1, 1>>>();
	if(hipSuccess != hipGetLastError())
	{
		return;
	}

	//wait for child to complete
	if(hipSuccess != hipDeviceSynchronize())
	{
		return;
	}

	printf("World!\n");
}

int main(int argc, char *argv[])
{
	//launch parent
	parentKernel<<<1, 1>>>();
	if(hipSuccess != hipGetLastError())
	{
		return 1;
	}

	//wait for parent to complete
	if(hipSuccess != hipDeviceSynchronize())
	{
		return 2;
	}

    return EXIT_SUCCESS;
}
