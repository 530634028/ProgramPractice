#include "hip/hip_runtime.h"
/****************************************
*
*  Program used for stream priority and
*  Multi-Device system.
*
*  a   : zhonghy
*  date: 2018-4-24
*****************************************/

//CUDA head files
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"       //maybe raise errors, because use v8.0 hip/hip_runtime_api.h?

#include <stdio.h>
#include <string>
#include <vector>
#include <iostream>


//hipError_t, remember
__global__ void MyKernel(float *p)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	p[tid] *= p[tid];  
}

int main(int argc, char *argv[])
{
   ////stream priorities
   ////get the range of stream priorities for this device
   //int priority_high, priority_low;
   //checkCudaErrors(hipDeviceGetStreamPriorityRange(&priority_high, &priority_low));
   ////create streams with highest and lowest available priorities
   //hipStream_t st_high, st_low;
   //checkCudaErrors(hipStreamCreateWithPriority(&st_high, 
	  // hipStreamNonBlocking, priority_high));
   //checkCudaErrors(hipStreamCreateWithPriority(&st_low,
	  // hipStreamNonBlocking, priority_low));
   //std::cout << priority_high << " " << priority_low << std::endl;

   //checkCudaErrors(hipStreamDestroy(st_high));
   //checkCudaErrors(hipStreamDestroy(st_low));


	/**********************************************************/
	//Multi-Device
	int deviceCount;
	checkCudaErrors(hipGetDeviceCount(&deviceCount));
	int device;
	for(device = 0; device < deviceCount; ++device)
	{
		hipDeviceProp_t deviceProp;
		checkCudaErrors(hipGetDeviceProperties(&deviceProp, device));
		std::cout << "Device " << device << " has compute capability "
			<< deviceProp.major << "." << deviceProp.minor << "." << std::endl;
	}

	size_t size = 1024 * sizeof(float);
	checkCudaErrors(hipSetDevice(0));            //set device 0 as current
	hipStream_t s0;
	checkCudaErrors(hipStreamCreate(&s0));
	float *p0;
	float *p0_h;
	p0_h = (float*)malloc(size);
	for(int i = 0; i < 1024; ++i)
	{
		p0_h[i] = i;
	}
	checkCudaErrors(hipMalloc(&p0, size));       //Allocate memeory on device 0
	checkCudaErrors(hipMemcpy(p0, p0_h, size, hipMemcpyHostToDevice));
	MyKernel<<<1000, 128, 0, s0>>>(p0); //Launch kernel on device 0
	checkCudaErrors(hipMemcpy(p0_h, p0, size, hipMemcpyDeviceToHost));

	//for p-p access
	checkCudaErrors(hipSetDevice(1));
	checkCudaErrors(hipDeviceEnablePeerAccess(0, 0));
	MyKernel<<<1000, 128>>>(p0); //Launch kernel on device 0

	for(int i = 0; i < 5; ++i)
	{
		std::cout << p0_h[i] << " ";
	}
	std::cout << std::endl;
	std::cout << hipGetLastError() << std::endl;



	////if have other device 1, for test
	checkCudaErrors(hipSetDevice(1));            //Set device 1 as current
	hipStream_t s1;
	checkCudaErrors(hipStreamCreate(&s1));
	float *p1;
	float *p1_h;
	p1_h = (float*)malloc(size);
	for(int i = 0; i < 1024; ++i)
	{
		p1_h[i] = i;
	}
	checkCudaErrors(hipMalloc(&p1, size));       //Allocate memeory on device 0
	checkCudaErrors(hipMemcpy(p1, p1_h, size, hipMemcpyHostToDevice));
	MyKernel<<<1000, 128, 0, s1>>>(p1); //Launch kernel on device 1

	//This kernel launch will fail: Launch kernel on device 1 in s0
	MyKernel<<<1000, 128, 0, s0>>>(p1);

	checkCudaErrors(hipMemcpy(p1_h, p1, size, hipMemcpyDeviceToHost));
	for(int i = 0; i < 5; ++i)
	{
		std::cout << p1_h[i] << " ";
	}
	std::cout << std::endl;

	checkCudaErrors(hipFree(p0));
	checkCudaErrors(hipFree(p1));
	free(p0_h);
	free(p1_h);
	checkCudaErrors(hipStreamDestroy(s0));
	checkCudaErrors(hipStreamDestroy(s1));

	/*****************************************************************/
	//p-p memory copy
	checkCudaErrors(hipSetDevice(0));   //set device 0 as current
	float *p2;
	size_t size1 = 1024 * sizeof(float);
	checkCudaErrors(hipMalloc(&p2, size));  //Allocate memory on device 0
	checkCudaErrors(hipSetDevice(1));       //set device 1 as current
	float *p3;
	checkCudaErrors(hipMalloc(&p3, size));  //Allocate memory on device 1
	checkCudaErrors(hipSetDevice(0));       //set device 0 as current
	MyKernel<<<1000, 128>>>(p2);             //Launch kernel on device 0

	checkCudaErrors(hipSetDevice(1));       //set device 1 as current
	checkCudaErrors(hipMemcpyPeer(p3, 1, p2, 0, size)); //Copy p2 to p3, memory copy from device to device
	MyKernel<<<1000, 128>>>(p3);             //Launch kernel on device 1

    return 0;
}
