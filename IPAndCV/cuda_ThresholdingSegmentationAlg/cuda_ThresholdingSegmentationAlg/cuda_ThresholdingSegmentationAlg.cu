#include "hip/hip_runtime.h"

/****************************************************
  Implementation for threshold segmentation algorithm
 with CUDA.
 a:    zhonghy
 date: 2019-4-10
*****************************************************/

#include "cuda_ThresholdingSegmentationAlg.h"

//const int thresold = 200;
//static __constant__ __device__ int conBallTable[520];
const int MaxValue = 255;
const int MinValue = 0;

// print necessary information into file, log_print or log_printf is already defined in CUDA???
void log_print(const char *filename, const char *str)   //__declspec(dllexport) 
{
	FILE *fp = fopen(filename,"a");//"log_gpu.txt"
	fprintf(fp,"%s",str);
	fclose(fp);
}

__global__ void cuda_ThresholdingSegmentationAlg_Kernel(const unsigned char *input, unsigned char *output,
	                                                    int imageW, int imageH, int thresold)
{
	int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
	int yIndex = threadIdx.y + blockIdx.y * blockDim.y;
	int dataIndex = yIndex * imageW + xIndex;
	if(input[dataIndex] < thresold)
	{
		output[dataIndex] = MinValue;
	}
	else
	{
		output[dataIndex] = MaxValue;
	}
}

int cpu_ThresholdingSegmentationAlg(const Mat &input, Mat &output, int thresold)
{
	if(input.empty())
	{
		return -1;
	}
	int rows = input.rows;
	int cols = input.cols * input.channels();
	for(int i = 0; i < rows; ++i)
	{
		const unsigned char *inputDataPtr = input.ptr<unsigned char>(i);  // image access method of opencv
		unsigned char *outputDataPtr = output.ptr<unsigned char>(i);
		for(int j = 0; j < cols; ++j)
		{
			if(inputDataPtr[j] < thresold)
			{
				outputDataPtr[j] = MinValue;
			}
			else
			{
				outputDataPtr[j] = MaxValue;
			}
		}
	}
	return 1;
}

// Implementation for threshold segmentation algorithm using CUDA 
hipError_t cuda_ThresholdingSegmentationAlg(const Mat &inputImage, Mat &outputImage, int thresold)
{
	unsigned char *dev_inputData = 0;
	unsigned char *dev_outputData = 0;
	int imageWidth = inputImage.cols;
	int imageHeight = inputImage.rows;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) 
	{
       fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
       return cudaStatus;
    }

	int size = imageWidth * imageHeight;
	cudaStatus = hipMalloc((void**)&dev_inputData, size * sizeof(unsigned char));
	cudaStatus = hipMalloc((void**)&dev_outputData, size * sizeof(unsigned char));

	cudaStatus = hipMemcpy(dev_inputData, inputImage.data, size * sizeof(unsigned char), hipMemcpyHostToDevice);
	cuda_ThresholdingSegmentationAlg_Kernel<<<size / 512, 512 >>>(dev_inputData, dev_outputData, imageWidth, imageHeight, thresold);
	cudaStatus = hipMemcpy(outputImage.data, dev_outputData, size * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

    hipFree(dev_inputData);
    hipFree(dev_outputData);

    return cudaStatus;
}
