#include "hip/hip_runtime.h"

/********************************************************
*
*
* This program is used to test Streams for CUDA performance
* improvement!
* a:zhonghy
*
*
*********************************************************/


//#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include <vector>
#include <string>
#include <stdio.h>
#include <math.h>
#include <iostream>

const int N = (1024*1024);
const int FULL_DATA_SIZE  = N * 20;



//for test kernel,c is output
__global__ void testKernel(int *c, const int *a, const int *b)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N)
	{
		c[idx] = (a[idx] + b[idx]) / 2;
	}
}

//unuse stream(default)
int UnUsedStreams()
{
	//start event clock
	hipEvent_t start, stop;
	float elapsedTime;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));

	int *host_a = 0, *host_b = 0, *host_c = 0;
	int *dev_a = 0, *dev_b = 0, *dev_c = 0;

	//alloc memory in GPU
	checkCudaErrors(hipMalloc((void **)&dev_a, FULL_DATA_SIZE * sizeof(int)));  //remember dev_a need &
	checkCudaErrors(hipMalloc((void **)&dev_b, FULL_DATA_SIZE * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&dev_c, FULL_DATA_SIZE * sizeof(int)));

	//alloc memory in CPU
	host_a = (int *)malloc(FULL_DATA_SIZE * sizeof(int));
	host_b = (int *)malloc(FULL_DATA_SIZE * sizeof(int));
	host_c = (int *)malloc(FULL_DATA_SIZE * sizeof(int));

	//assign vlaues in cpu
	for(int i = 0; i < FULL_DATA_SIZE; ++i)
	{
		host_a[i] = i;
		host_b[i] = FULL_DATA_SIZE -i;
	}

	//copy data from CPU to GPU
	checkCudaErrors(hipMemcpy(dev_a, host_a, FULL_DATA_SIZE * sizeof(int),
		hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_b, host_b, FULL_DATA_SIZE * sizeof(int),
		hipMemcpyHostToDevice));

	testKernel<<<FULL_DATA_SIZE / 1024, 1024>>>(dev_c, dev_a, dev_b);

	checkCudaErrors(hipMemcpy(host_c, dev_c, FULL_DATA_SIZE * sizeof(int),
		hipMemcpyDeviceToHost));

	//end of clock
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));

	std::cout << "Time consume: " << elapsedTime << std::endl;

	//show output
	for(int i = 0; i < 10; ++i)
	{
		std::cout << host_c[i] << std::endl;
	}

	//getchar();


	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	return 0;
}

int UsedStreams()
{
	//attain device properties
	hipDeviceProp_t prop;
	int deviceID;
	checkCudaErrors(hipGetDevice(&deviceID));
	checkCudaErrors(hipGetDeviceProperties(&prop, deviceID));

	//test if have overlap
	if(!prop.deviceOverlap)
	{
		//just write like endl, get error MSB3721: The command ""C:\Program Files\NVIDIA GPU Computing
		std::cout << "No device will handle overlaps." << std::endl; 
		return 0;
	}

	//start event clock
	hipEvent_t start, stop;
	float elapsedTime;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));

	//create streams
	hipStream_t stream;
	hipStreamCreate(&stream);

	int *host_a = 0, *host_b = 0, *host_c = 0;
	int *dev_a = 0, *dev_b = 0, *dev_c = 0;

	//alloc memory in GPU
	checkCudaErrors(hipMalloc((void **)&dev_a, N * sizeof(int))); //remember dev_a need &
	checkCudaErrors(hipMalloc((void **)&dev_b, N * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&dev_c, N * sizeof(int)));

	//alloc memory in CPU, must pined memory
	checkCudaErrors(hipHostAlloc((void **)&host_a, FULL_DATA_SIZE * sizeof(int),
		hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void **)&host_b, FULL_DATA_SIZE * sizeof(int),
		hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void **)&host_c, FULL_DATA_SIZE * sizeof(int),
		hipHostMallocDefault));

	//assign vlaues in cpu
	for(int i = 0; i < FULL_DATA_SIZE; ++i)
	{
		host_a[i] = i;
		host_b[i] = FULL_DATA_SIZE - i;
	}

	for(int i= 0; i < FULL_DATA_SIZE; i += N)
	{
		checkCudaErrors(hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int),
			hipMemcpyHostToDevice, stream));
		checkCudaErrors(hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int),
			hipMemcpyHostToDevice, stream));

		testKernel<<<N / 1024, 1024, 0, stream>>>(dev_c, dev_a, dev_b);

		checkCudaErrors(hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int),
			hipMemcpyDeviceToHost, stream));
	}

	//wait until gpu execution finish
	checkCudaErrors(hipStreamSynchronize(stream));  //later add 


	//end of clock
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));

	std::cout << "Time consume: " << elapsedTime << std::endl;

	//show output
	for(int i = 0; i < 10; ++i)
	{
		std::cout << host_c[i] << std::endl;
	}

	//getchar();

	checkCudaErrors(hipHostFree(host_a));
	checkCudaErrors(hipHostFree(host_b));
	checkCudaErrors(hipHostFree(host_c));

	checkCudaErrors(hipFree(dev_a));
	checkCudaErrors(hipFree(dev_b));
	checkCudaErrors(hipFree(dev_c));

	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	return 0;
}


int main(int argc, char *argv[])
{
    UnUsedStreams(); 
	UsedStreams();//why cann't runs, because of the getchar() function
    return 0;
}