#include "hip/hip_runtime.h"

/********************************************************
*
*
* This program is used to test Streams for CUDA performance
* improvement!
* a:zhonghy
*
*
*********************************************************/


//#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include <vector>
#include <string>
#include <stdio.h>
#include <math.h>

const int N = (1024*1024);
const int FULL_DATA_SIZE  = N * 20;



//for test kernel,c is output
__global__ void testKernel(int *c, const int *a, const int *b)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N)
	{
		c[idx] = (a[idx] + b[idx]) / 2;
	}
}

//unuse stream(default)
void UnUsedStreams()
{
	//start event clock
	hipEvent_t start, stop;
	float elapsedTime;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));

	int *host_a, *host_b, *host_c;
	int *dev_a, *dev_b, *dev_c;

	//alloc memory in GPU
	checkCudaErrors(hipMalloc((void **)dev_a, FULL_DATA_SIZE * sizeof(int)));
	checkCudaErrors(hipMalloc((void **)dev_b, FULL_DATA_SIZE * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)dev_c, FULL_DATA_SIZE * sizeof(int)));

	//alloc memory in CPU
	host_a = (int *)malloc(FULL_DATA_SIZE * sizeof(int));
	host_b = (int *)malloc(FULL_DATA_SIZE * sizeof(int));
	host_c = (int *)malloc(FULL_DATA_SIZE * sizeof(int));

	//assign vlaues in cpu
	for(int i = 0; i < FULL_DATA_SIZE; ++i)
	{
		host_a[i] = i;
		host_b[i] = FULL_DATA_SIZE -i;
	}

	//copy data from CPU to GPU
	checkCudaErrors(hipMemcpy(dev_a, host_a, FULL_DATA_SIZE * sizeof(int),
		hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_b, host_b, FULL_DATA_SIZE * sizeof(int),
		hipMemcpyHostToDevice));

	testKernel<<<FULL_DATA_SIZE / 1024, 1024>>>(dev_c, dev_a, dev_b);

	checkCudaErrors(hipMemcpy(host_c, dev_c, FULL_DATA_SIZE * sizeof(int),
		hipMemcpyDeviceToHost));

	//end of clock
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));

	std::cout << "Time consume: " << elapsedTime << std::endl;

	//show output
	for(int i = 0; i < 10; ++i)
	{
		std::cout << host_c[i] << std::endl;
	}

	getchar();


	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));
}

int UsedStreams()
{
	//attain device properties
	hipDeviceProp_t prop;
	int deviceID;
	checkCudaErrors(hipGetDevice(&deviceID));
	checkCudaErrors(hipGetDeviceProperties(&prop, deviceID));

	//test if have overlap
	if(!prop.deviceOverlap)
	{
		std::cout << "No device will handle overlaps." << endl;
		return 0;
	}

	//start event clock
	hipEvent_t start, stop;
	float elapsedTime;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));

	//create streams
	hipStream_t stream;
	hipStreamCreate(&stream);

	int *host_a, *host_b, *host_c;
	int *dev_a, *dev_b, *dev_c;

	//alloc memory in GPU
	checkCudaErrors(hipMalloc((void **)dev_a, N * sizeof(int)));
	checkCudaErrors(hipMalloc((void **)dev_b, N * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)dev_c, N * sizeof(int)));

	//alloc memory in CPU, must pined memory
	checkCudaErrors(hipHostAlloc((void **)&host_a, FULL_DATA_SIZE * sizeof(int),
		hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void **)&host_b, FULL_DATA_SIZE * sizeof(int),
		hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void **)&host_c, FULL_DATA_SIZE * sizeof(int),
		hipHostMallocDefault));

	//assign vlaues in cpu
	for(int i = 0; i < FULL_DATA_SIZE; ++i)
	{
		host_a[i] = i;
		host_b[i] = FULL_DATA_SIZE -i;
	}

	for(int i= 0; i < FULL_DATA_SIZE; i +=N)
	{
		checkCudaErrors(hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int),
			hipMemcpyHostToDevice, stream));
		checkCudaErrors(hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int),
			hipMemcpyHostToDevice, stream));

		testKernel<<<N /1024, 1024, 0, stream>>>(dev_c, dev_a, dev_b);

		checkCudaErrors(hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int),
			hipMemcpyDeviceToHost, stream));
	}

	//end of clock
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));

	std::cout << "Time consume: " << elapsedTime << std::endl;

	//show output
	for(int i = 0; i < 10; ++i)
	{
		std::cout << host_c[i] << std::endl;
	}

	getchar();

	checkCudaErrors(hipHostFree(host_a));
	checkCudaErrors(hipHostFree(host_b));
	checkCudaErrors(hipHostFree(host_c));

	checkCudaErrors(hipFree(dev_a));
	checkCudaErrors(hipFree(dev_b));
	checkCudaErrors(hipFree(dev_c));

	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	return 0;
}


int main(int argc, char *argv[])
{
    UnUsedStreams();
	UsedStreams();
    return 0;
}