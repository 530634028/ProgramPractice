#include "hip/hip_runtime.h"

/********************************************************
*
*
* This program is used to test Streams for CUDA performance
* improvement!
* a:zhonghy
*
*
*********************************************************/


//#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include <vector>
#include <string>
#include <stdio.h>
#include <math.h>
#include <iostream>

const int N = (1024*1024);
const int FULL_DATA_SIZE  = N * 20;



//for test kernel,c is output
__global__ void testKernel(int *c, const int *a, const int *b)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N)
	{
		c[idx] = (a[idx] + b[idx]) / 2;
	}
}

//add callback functions 2018-4-23 zhong
void CUDART_CB MyCallback(hipStream_t stream, hipError_t status, void *data)
{
	std::cout << "Inside callback " << (size_t)data << std::endl;
}

//unuse stream(default)
int UnUsedStreams()
{
	//start event clock
	hipEvent_t start, stop;
	float elapsedTime;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));

	int *host_a = 0, *host_b = 0, *host_c = 0;
	int *dev_a = 0, *dev_b = 0, *dev_c = 0;

	//alloc memory in GPU
	checkCudaErrors(hipMalloc((void **)&dev_a, FULL_DATA_SIZE * sizeof(int)));  //remember dev_a need &
	checkCudaErrors(hipMalloc((void **)&dev_b, FULL_DATA_SIZE * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&dev_c, FULL_DATA_SIZE * sizeof(int)));

	//alloc memory in CPU
	host_a = (int *)malloc(FULL_DATA_SIZE * sizeof(int));
	host_b = (int *)malloc(FULL_DATA_SIZE * sizeof(int));
	host_c = (int *)malloc(FULL_DATA_SIZE * sizeof(int));

	//assign vlaues in cpu
	for(int i = 0; i < FULL_DATA_SIZE; ++i)
	{
		host_a[i] = i;
		host_b[i] = FULL_DATA_SIZE -i;
	}

	//copy data from CPU to GPU
	checkCudaErrors(hipMemcpy(dev_a, host_a, FULL_DATA_SIZE * sizeof(int),
		hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_b, host_b, FULL_DATA_SIZE * sizeof(int),
		hipMemcpyHostToDevice));

	testKernel<<<FULL_DATA_SIZE / 1024, 1024>>>(dev_c, dev_a, dev_b);

	checkCudaErrors(hipMemcpy(host_c, dev_c, FULL_DATA_SIZE * sizeof(int),
		hipMemcpyDeviceToHost));

	//end of clock
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));

	std::cout << "UnUsedStreams Time consume: " << elapsedTime << std::endl;

	//show output
	for(int i = 0; i < 10; ++i)
	{
		std::cout << host_c[i] << std::endl;
	}

	//getchar();


	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	return 0;
}

int UsedStreams()
{
	//attain device properties
	hipDeviceProp_t prop;
	int deviceID;
	checkCudaErrors(hipGetDevice(&deviceID));
	checkCudaErrors(hipGetDeviceProperties(&prop, deviceID));

	//test if have overlap
	if(!prop.deviceOverlap)
	{
		//just write like endl, get error MSB3721: The command ""C:\Program Files\NVIDIA GPU Computing
		std::cout << "No device will handle overlaps." << std::endl; 
		return 0;
	}

	//start event clock
	hipEvent_t start, stop;
	float elapsedTime;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));

	//create streams
	hipStream_t stream;
	hipStreamCreate(&stream);

	int *host_a = 0, *host_b = 0, *host_c = 0;
	int *dev_a = 0, *dev_b = 0, *dev_c = 0;

	//alloc memory in GPU
	checkCudaErrors(hipMalloc((void **)&dev_a, N * sizeof(int))); //remember dev_a need &
	checkCudaErrors(hipMalloc((void **)&dev_b, N * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&dev_c, N * sizeof(int)));

	//alloc memory in CPU, must pined memory
	checkCudaErrors(hipHostAlloc((void **)&host_a, FULL_DATA_SIZE * sizeof(int),
		hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void **)&host_b, FULL_DATA_SIZE * sizeof(int),
		hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void **)&host_c, FULL_DATA_SIZE * sizeof(int),
		hipHostMallocDefault));

	//assign vlaues in cpu
	for(int i = 0; i < FULL_DATA_SIZE; ++i)
	{
		host_a[i] = i;
		host_b[i] = FULL_DATA_SIZE - i;
	}

	for(int i= 0; i < FULL_DATA_SIZE; i += N)
	{
		checkCudaErrors(hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int),
			hipMemcpyHostToDevice, stream));
		checkCudaErrors(hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int),
			hipMemcpyHostToDevice, stream));

		testKernel<<<N / 1024, 1024, 0, stream>>>(dev_c, dev_a, dev_b);

		checkCudaErrors(hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int),
			hipMemcpyDeviceToHost, stream));
	}

	//wait until gpu execution finish
	checkCudaErrors(hipStreamSynchronize(stream));  //later add 


	//end of clock
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));

	std::cout << "UsedStreams Time consume: " << elapsedTime << std::endl;

	//show output
	for(int i = 0; i < 10; ++i)
	{
		std::cout << host_c[i] << std::endl;
	}

	//getchar();

	checkCudaErrors(hipHostFree(host_a));
	checkCudaErrors(hipHostFree(host_b));
	checkCudaErrors(hipHostFree(host_c));

	checkCudaErrors(hipFree(dev_a));
	checkCudaErrors(hipFree(dev_b));
	checkCudaErrors(hipFree(dev_c));

	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	return 0;
}

//staged concurrent copy and execute
int UsedStagedStreams() //something wrong, why faster than UsedStreams
{
	//std::cout << "Used Staged Strems: " << std::endl; 
	//attain device properties
	const int nStreams = 8; //number of used non-null stream
	hipDeviceProp_t prop;
	int deviceID;
	checkCudaErrors(hipGetDevice(&deviceID));
	checkCudaErrors(hipGetDeviceProperties(&prop, deviceID));

	//test if have overlap
	if(!prop.deviceOverlap)
	{
		//just write like endl, get error MSB3721: The command ""C:\Program Files\NVIDIA GPU Computing
		std::cout << "No device will handle overlaps." << std::endl; 
		return 0;
	}

	//start event clock
	hipEvent_t start, stop;
	float elapsedTime;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));

	//create streams
	//hipStream_t stream;  //error arise--repetition next
	//hipStreamCreate(&stream);

	int *host_a = 0, *host_b = 0, *host_c = 0;
	int *dev_a = 0, *dev_b = 0, *dev_c = 0;


	//alloc memory in GPU
	checkCudaErrors(hipMalloc((void **)&dev_a, FULL_DATA_SIZE * sizeof(int))); //remember dev_a need &
	checkCudaErrors(hipMalloc((void **)&dev_b, FULL_DATA_SIZE * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&dev_c, FULL_DATA_SIZE * sizeof(int)));

	//alloc memory in CPU, must pined memory
	checkCudaErrors(hipHostAlloc((void **)&host_a, FULL_DATA_SIZE * sizeof(int),
		hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void **)&host_b, FULL_DATA_SIZE * sizeof(int),
		hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc((void **)&host_c, FULL_DATA_SIZE * sizeof(int),
		hipHostMallocDefault));

	//assign vlaues in cpu
	for(int i = 0; i < FULL_DATA_SIZE; ++i)
	{
		host_a[i] = i;
		host_b[i] = FULL_DATA_SIZE - i;
	}


	int sizeOfStream = FULL_DATA_SIZE * sizeof(int) / nStreams;

	hipStream_t stream[nStreams];
	for(int i = 0; i < nStreams; i++)
	{
		checkCudaErrors(hipStreamCreate(&stream[i]));
	}


	////add callback functions 
	//for(int i= 0; i < nStreams; i++)
	//{
	//	int offset = i * FULL_DATA_SIZE / nStreams; //it is different from sizeOfStream
	//	checkCudaErrors(hipMemcpyAsync(dev_a + offset, host_a + offset, sizeOfStream,
	//		hipMemcpyHostToDevice, stream[i]));
	//	checkCudaErrors(hipMemcpyAsync(dev_b + offset, host_b + offset, sizeOfStream,
	//		hipMemcpyHostToDevice, stream[i]));

	//	testKernel<<<FULL_DATA_SIZE / (nStreams * 1024) , 1024, 0, stream[i]>>>(dev_c + offset, dev_a + offset, dev_b + offset);

	//	checkCudaErrors(hipMemcpyAsync(host_c + offset, dev_c + offset, sizeOfStream,
	//		hipMemcpyDeviceToHost, stream[i]));

	//	//add callback functions
	//	checkCudaErrors(hipStreamAddCallback(stream[i], MyCallback, (void*)i, 0));
	//}

	//another way to run zhonghy-2018-4-19 added, in 3.2.5.5 of guide
	for(int i= 0; i < nStreams; i++)
	{
		int offset = i * FULL_DATA_SIZE / nStreams; //it is different from sizeOfStream
		checkCudaErrors(hipMemcpyAsync(dev_a + offset, host_a + offset, sizeOfStream,
			hipMemcpyHostToDevice, stream[i]));
	}

	for(int i= 0; i < nStreams; i++)
	{
		int offset = i * FULL_DATA_SIZE / nStreams; //it is different from sizeOfStream
		checkCudaErrors(hipMemcpyAsync(dev_b + offset, host_b + offset, sizeOfStream,
			hipMemcpyHostToDevice, stream[i]));
	}

	for(int i= 0; i < nStreams; i++)
	{
		int offset = i * FULL_DATA_SIZE / nStreams; //it is different from sizeOfStream
		testKernel<<<FULL_DATA_SIZE / (nStreams * 1024) , 1024, 0, stream[i]>>>(dev_c + offset, dev_a + offset, dev_b + offset);
	}

	for(int i= 0; i < nStreams; i++)
	{
		int offset = i * FULL_DATA_SIZE / nStreams; //it is different from sizeOfStream
		checkCudaErrors(hipMemcpyAsync(host_c + offset, dev_c + offset, sizeOfStream,
			hipMemcpyDeviceToHost, stream[i]));
	}
	


	for(int i = 0; i < nStreams; ++i)
	{
				//wait until gpu execution finish
		checkCudaErrors(hipStreamSynchronize(stream[i]));  //later add 
	}

   
	//end of clock
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));

	std::cout << "UsedStagedStreams Time consume: " << elapsedTime << std::endl;

	//show output
	for(int i = 0; i < 10; ++i)
	{
		std::cout << host_c[i] << std::endl;
	}

	//getchar();

	for(int i = 0; i < nStreams; ++i)
	{
		//wait until gpu execution finish
		checkCudaErrors(hipStreamDestroy(stream[i]));  //later add 
	}

	//free all memory 
	checkCudaErrors(hipHostFree(host_a));
	checkCudaErrors(hipHostFree(host_b));
	checkCudaErrors(hipHostFree(host_c));

	checkCudaErrors(hipFree(dev_a));
	checkCudaErrors(hipFree(dev_b));
	checkCudaErrors(hipFree(dev_c));

	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	return 0;
}

int main(int argc, char *argv[])
{
    UnUsedStreams(); // nonStreams
	UsedStreams();//20 streams
	UsedStagedStreams();//4 Streams
	//why cann't runs, because of the getchar() function
    return 0;
}