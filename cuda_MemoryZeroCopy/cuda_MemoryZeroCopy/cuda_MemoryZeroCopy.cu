#include "hip/hip_runtime.h"

/********************************************
*
* Usage: Test for hipHostAlloc() function
* a: zhy
*
* https://www.cnblogs.com/zhangshuwen/p/7349267.html
*
*********************************************/

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include <stdio.h>

#define imin(a, b) (a<b?a:b);

const int N = 33 * 1024 *1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void dot(int size, float *a, float *b, float *c)
{
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;

	float temp = 0;
	while(tid < size)
	{
		temp += a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	}

	//set the cache values
	cache[cacheIndex] = temp;

	//synchronize threads in this block
	__syncthreads();

    //for reductions, threadsPerBlock must be a power of 2
	//because of the following code
	int i = blockDim.x /2;
	while(i != 0)
	{
		if(cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];
		__syncthreads();
		i /= 2;
	}

	if(cacheIndex == 0)
		c[blockIdx.x] = cache[0];
}


//hipMalloc memory version
float malloc_test(int size)
{
	hipEvent_t start, stop;
	float *a, *b, c, *partial_c;
	float *dev_a, *dev_b, *dev_partial_c;
	float elapsedTime;

	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	//alloc in cpu memory
	a = (float *)malloc(size * sizeof(float));
	b = (float *)malloc(size * sizeof(float));
	partial_c = (float *)malloc(blocksPerGrid * sizeof(float));
	
	//alloc in GPU memory
	checkCudaErrors(hipMalloc((void **)&dev_a, size * sizeof(float)));
	checkCudaErrors(hipMalloc((void **)&dev_b, size * sizeof(float)));
	checkCudaErrors(hipMalloc((void **)&dev_partial_c, blocksPerGrid * sizeof(float)));

	//filling CPU memory alloced with data
	for(int i = 0; i < size; ++i)
	{
		a[i] = i;
		b[i] = i * 2;
	}

	checkCudaErrors(hipEventRecord(start, 0));
	checkCudaErrors(hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice));

	dot<<<blocksPerGrid, threadsPerBlock>>>(size, dev_a, dev_b, dev_partial_c);

	//copy data from GPU to CPU
	checkCudaErrors(hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));

	//exit cal in cpu
	c = 0;
	for(int i = 0; i < blocksPerGrid; i++)
	{
		c += partial_c[i];
	}

	//delete pointer
	checkCudaErrors(hipFree(dev_a));
	checkCudaErrors(hipFree(dev_b));
	checkCudaErrors(hipFree(dev_partial_c));

	free(a);
	free(b);
	free(partial_c);

	//release the event
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	printf("Value calculated: %f\n", c);

	return elapsedTime;
}


//zero copy version
float cuda_host_alloc_test(int size)
{
	hipEvent_t start, stop;
	float *a, *b, c, *partial_c;
	float *dev_a, *dev_b, *dev_partial_c;
	float elapsedTime;

	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	//alloc in cpu memory
	checkCudaErrors(hipHostAlloc((void **)&a, size * sizeof(float), //hipHostAlloc();
		hipHostMallocWriteCombined | hipHostMallocMapped));
	checkCudaErrors(hipHostAlloc((void **)&b, size * sizeof(float),
		hipHostMallocWriteCombined | hipHostMallocMapped));
	checkCudaErrors(hipHostAlloc((void **)&partial_c, blocksPerGrid * sizeof(float),
		hipHostMallocWriteCombined | hipHostMallocMapped));

	//filling CPU memory alloced with data
	for(int i = 0; i < size; ++i)
	{
		a[i] = i;
		b[i] = i * 2;
	}

	checkCudaErrors(hipHostGetDevicePointer(&dev_a, a, 0));
	checkCudaErrors(hipHostGetDevicePointer(&dev_b, b, 0));               //hipHostGetDevicePointer()
	checkCudaErrors(hipHostGetDevicePointer(&dev_partial_c, partial_c, 0));

	for(int i = 0; i < size; i++)
	{
		a[i] = i;
		b[i] = i * 2;
	}

	checkCudaErrors(hipEventRecord(start, 0));

	dot<<<blocksPerGrid, threadsPerBlock>>>(size, dev_a, dev_b, dev_partial_c);

	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));

	//exit cal in cpu
	c = 0;
	for(int i = 0; i < blocksPerGrid; i++)
	{
		c += partial_c[i];
	}

	//free pointer
	checkCudaErrors(hipHostFree(a));    //hipHostFree()
	checkCudaErrors(hipHostFree(b));
	checkCudaErrors(hipHostFree(partial_c));

	//free the event
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	printf("Value calculated: %f\n", c);

	return elapsedTime;
}


int main(int argc, char *argv[])
{
   hipDeviceProp_t prop;
   int whichDevice;
   checkCudaErrors(hipGetDevice(&whichDevice));
   checkCudaErrors(hipGetDeviceProperties(&prop, whichDevice));

   if(prop.canMapHostMemory != 1)
   {
	   printf("Device can not map memory.\n");
	   return 0;
   }

   float elapsedTime;
   checkCudaErrors(hipSetDeviceFlags(hipDeviceMapHost));
    
   //try it with malloc
   elapsedTime = malloc_test(N);
   printf("Time using hipMalloc:   %3.lf ms\n", elapsedTime);


   //try it with hipHostAlloc
   elapsedTime = cuda_host_alloc_test(N);
   printf("Time using hipHostAlloc:  %3.lf ms\n", elapsedTime);

}
