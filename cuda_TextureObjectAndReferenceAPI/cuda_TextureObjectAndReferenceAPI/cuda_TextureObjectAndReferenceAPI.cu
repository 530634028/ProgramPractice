#include "hip/hip_runtime.h"

/******************************************************
*
* Program used to test texture object and reference API
*
* a   : zhonghy
* date: 2018-4-24
********************************************************/


#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include <iostream>
#include <string>
#include <vector>
#include <stdio.h>



//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


//Simple transformtion kernel
__global__ void transformKernel(float *output, hipTextureObject_t texObj,
	                            int width, int height, float theta)
{
	//Calculate normalized texture coordinates
	//2D block
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	float u = x / (float)width;
	float v = y / (float)height;

	//transform coordinates
	u -= 0.5f;
	v -= 0.5f;
	float tu = u * cosf(theta) - v * sinf(theta) + 0.5f;
	float tv = v * cosf(theta) + u * sinf(theta) + 0.5f;

	//Read from texture and write to global memory
	output[y * width + x] = tex2D<float>(texObj, tu, tv);
}


int main(int argc, char *argv[])
{
	//Allocate CUDA array in device memory
	int width = 1024;
	int height = 1024;
	int dataSize = width * height;
	float *h_data;
	h_data = (float*)malloc(dataSize * sizeof(float));


	hipChannelFormatDesc channelDesc = 
		hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipArray *cuArray;
	checkCudaErrors(hipMallocArray(&cuArray, &channelDesc, width, height));

	//Copy to device memory some data located at address h_data
	//in host memory
	checkCudaErrors(hipMemcpyToArray(hipArray, 0, 0, h_data,
		size, hipMemcpyHostToDevice));

	//Specify texture
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;

	//Specify texture object parameters
	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0]   = hipAddressModeWrap;
	texDesc.addressMode[1]   = hipAddressModeWrap;
	texDesc.filterMode       = hipFilterModeLinear;
	texDesc.readMode         = hipReadModeElementType;
	texDesc.normalizedCoords = 1;

	//Create texture object
	hipTextureObject_t texObj = 0;
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

	//Allocate result of transformation in device memory
	float *output;
	checkCudaErrors(hipMalloc(&output, width * height * sizeof(float)));

	//Invoke kernel
	dim3 dimBlock(16, 16);
	dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, 
		         (height + dimBlock.y - 1) / dimBlock.y);
	transformKernel<<<dimGrid, dimBlock>>>(output, 
		      texObj, width, height, angle);

	//Destory texture object
	hipDestroyTextureObject(texObj);

	//Free device memory
    hipFreeArray(cuArray);
    hipFree(output);
    
    return EXIT_SUCCESS;
}
