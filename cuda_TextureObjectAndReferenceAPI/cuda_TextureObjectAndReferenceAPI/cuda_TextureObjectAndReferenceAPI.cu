#include "hip/hip_runtime.h"

/******************************************************
*
* Program used to test texture object and reference API
*
* a   : zhonghy
* date: 2018-4-24
********************************************************/


#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include <iostream>
#include <string>
#include <vector>
#include <stdio.h>



//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
int print(float *, int, int);
__global__ void transformKernel(float *, hipTextureObject_t,
	                            int, int, float);

//2D float texture
texture<float, hipTextureType2D, hipReadModeElementType> texRef;

int print(float *mat, int width, int height)
{
	if(!mat)
	{
		return 0;
	}
	for(int i = 0; i < width; ++i)
	{
		for(int j = 0; j < height; ++j)
		{
			std::cout << mat[j * width + i] << " ";
		}
		std::cout << std::endl;
	}

}

//Simple transformation kernel
__global__ void transformKernel(float *output, hipTextureObject_t texObj,
	                            int width, int height, float theta)
{
	//Calculate normalized texture coordinates
	//2D block
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	float u = x / (float)width;
	float v = y / (float)height;

	//transform coordinates
	u -= 0.5f;
	v -= 0.5f;
	float tu = u * cosf(theta) - v * sinf(theta) + 0.5f;
	float tv = v * cosf(theta) + u * sinf(theta) + 0.5f;

	//Read from texture and write to global memory
	output[y * width + x] = tex2D<float>(texObj, tu, tv);
}

//Simple transfromation kernel
__global__ void transformKernelRef(float *output, int width,
	                               int height, float theta)
{
	//Calculate normalized texture coordinates
	//2D block
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	float u = x / (float)width;
	float v = y / (float)height;

	//transform coordinates
	u -= 0.5f;
	v -= 0.5f;
	float tu = u * cosf(theta) - v * sinf(theta) + 0.5f;
	float tv = v * cosf(theta) + u * sinf(theta) + 0.5f;

	//Read from texture and write to global memory
	output[y * width + x] = tex2D<float>(texRef, tu, tv);
}


int main(int argc, char *argv[])
{
	/**********************texture object*********************/
	////Allocate CUDA array in device memory
	//int width = 256;
	//int height = 256;
	//int angle = 30;
	//int size = width * height;
	//float *h_data;
	//h_data = (float*)malloc(size * sizeof(float));
	//for(int i = 0; i < width; ++i)
	//{
	//	for(int j = 0; j < height; ++j)
	//	{
	//		h_data[j * width + i] = (i + j + 2) / 2;
	//	}
	//}
	//print(h_data, 5 ,5);


	//hipChannelFormatDesc channelDesc = 
	//	hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	//hipArray *cuArray;
	//checkCudaErrors(hipMallocArray(&cuArray, &channelDesc, width, height));

	////Copy to device memory some data located at address h_data
	////in host memory
	//checkCudaErrors(hipMemcpyToArray(cuArray, 0, 0, h_data,
	//	size, hipMemcpyHostToDevice));

	////Specify texture
	//struct hipResourceDesc resDesc;
	//memset(&resDesc, 0, sizeof(resDesc));
	//resDesc.resType = hipResourceTypeArray;
	//resDesc.res.array.array = cuArray;

	////Specify texture object parameters
	//struct hipTextureDesc texDesc;
	//memset(&texDesc, 0, sizeof(texDesc));
	//texDesc.addressMode[0]   = hipAddressModeWrap;
	//texDesc.addressMode[1]   = hipAddressModeWrap;
	//texDesc.filterMode       = hipFilterModeLinear;
	//texDesc.readMode         = hipReadModeElementType;
	//texDesc.normalizedCoords = 1;

	////Create texture object
	//hipTextureObject_t texObj = 0;
	//hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

	////Allocate result of transformation in device memory
	//float *output;
	//checkCudaErrors(hipMalloc(&output, width * height * sizeof(float)));

	////Invoke kernel
	//dim3 dimBlock(16, 16);
	//dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, 
	//	         (height + dimBlock.y - 1) / dimBlock.y);
	//transformKernel<<<dimGrid, dimBlock>>>(output, 
	//	      texObj, width, height, angle);


	////read result from device
	//float *output_h;
	////output_h = (float*)malloc(size * sizeof(float));
	//output_h = new float[size];
	//checkCudaErrors(hipMemcpy(output_h, output, size, hipMemcpyDeviceToHost));
 //   print(output_h, 256, 10);

	////Destory texture object
	//hipDestroyTextureObject(texObj);

	////Free device memory
 //   hipFreeArray(cuArray);
 //   hipFree(output);
	//free(h_data);
	//delete[] output_h;
    
	/*********************texture reference*********************/
	//Allocate CUDA array in device memory
	int width = 256;
	int height = 256;
	int angle = 30;
	int size = width * height;
	float *h_data;
	h_data = (float*)malloc(size * sizeof(float));
	for(int i = 0; i < width; ++i)
	{
		for(int j = 0; j < height; ++j)
		{
			h_data[j * width + i] = (i + j + 2) / 2;
		}
	}
	print(h_data, 5 ,5);

	hipChannelFormatDesc channelDesc = 
		hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipArray *cuArray;
	checkCudaErrors(hipMallocArray(&cuArray, &channelDesc, width, height));

	//Copy to device memory some data located at address h_data
	//in host memory
	checkCudaErrors(hipMemcpyToArray(cuArray, 0, 0, h_data,
		size, hipMemcpyHostToDevice));

	//Specify texture object parameters
	texRef.addressMode[0]   = hipAddressModeWrap;
	texRef.addressMode[1]   = hipAddressModeWrap;
	texRef.filterMode       = hipFilterModeLinear;
	texRef.normalized       = 1;

	//Bind the array to the texture reference
    checkCudaErrors(hipBindTextureToArray(texRef, cuArray, channelDesc));

	//Allocate result of transformation in device memory
	float *output;
	checkCudaErrors(hipMalloc(&output, width * height * sizeof(float)));

	//Invoke kernel
	dim3 dimBlock(16, 16);
	dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, 
		         (height + dimBlock.y - 1) / dimBlock.y);
	transformKernelRef<<<dimGrid, dimBlock>>>(output, width, height, angle);


	//read result from device
	float *output_h;
	//output_h = (float*)malloc(size * sizeof(float));
	output_h = new float[size];
	checkCudaErrors(hipMemcpy(output_h, output, size, hipMemcpyDeviceToHost));
    print(output_h, 256, 10);

	//Free device memory
    hipFreeArray(cuArray);
    hipFree(output);
	free(h_data);
	delete[] output_h;

    return EXIT_SUCCESS;
}
