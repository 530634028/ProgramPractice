#include "hip/hip_runtime.h"
/***********************************************
*
*  This program is used for vector add 
*  date:2018-5-2
*  a:zhonghy
**********************************************/


#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"


#include <stdio.h>
#include <iostream>
#include <string>

//?
//#include <malloc.h>

const int N = 32;

//hipError_t addWithCuda1D(int *c, const int *a, const int *b, unsigned int size);
//hipError_t addWithCuda1D2D(int c[N][N], const int a[N][N], const int b[N][N], unsigned int size);
//hipError_t addWithCuda2D2D(int c[N][N], const int a[N][N], const int b[N][N], unsigned int size);

////1D blocksPerGrid 1D threadsPerBlock
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

//1D blocksPerGrid 2D threadsPerBlock matrix add
__global__ void MatAdd(int *C, int *A, int *B)   //wrong here C, A, B write to A B C
{
	int i = threadIdx.x;
	int j = threadIdx.y;

	//int index = blockDim.x * threadIdx.y + threadIdx.x;
	//C[index] = A[index] + B[index];    //here wrong write C[j * blockDim.x + i] = B[j * blockDim.x + i] + C(need to mend A)[j * blockDim.x + i];

	C[j * blockDim.x + i] = A[j * blockDim.x + i] + B[j * blockDim.x + i];
}

//2D blocksPerGrid 2D threadsPerBlock matrix add
__global__ void MatAdd_2D(int *C, int *A, int *B)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	C[j * blockDim.x * gridDim.x + i] = B[j * blockDim.x * gridDim.x + i] + A[j * blockDim.x * gridDim.x + i];
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda1D(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    //// Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda1D2D(int *c, int *a, int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

//#ifndef DEBUG
//	for(int i = 0; i < 5; ++i)
//	{
//		std::cout << b[i] << " ";
//	}
//
//#endif

    //// Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


	int numBlocks = 1;
	dim3 threadsPerBlock(32, 32);
    // Launch a kernel on the GPU with one thread for each element.
    MatAdd<<<numBlocks, threadsPerBlock>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	//for(int i = 0; i < 5; ++i)
	//{
	//	std::cout << c[i] << " ";
	//}


Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;

}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda2D2D(int *c, int *a, int *b, unsigned int size)
{
    int *dev_a = NULL;
    int *dev_b = NULL;
    int *dev_c = NULL;
    hipError_t cudaStatus;

    //// Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int ));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);
    // Launch a kernel on the GPU with one thread for each element.
    MatAdd_2D<<<numBlocks, threadsPerBlock>>>(dev_c, dev_a, dev_b);


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;

}

int main(int argc, char *argv[])
{
	//For 1D vector add
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda1D(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	std::cout << std::endl;



	/******************************************************/
	//For 2D matrix add
	int *A; 
	int *B;
	int *C;
	A = (int*)malloc(N * N * sizeof(int)); 
	B = (int*)malloc(N * N * sizeof(int));
	C = (int*)malloc(N * N * sizeof(int));

	//for(int i = 0; i < N; ++i)
	//{
	//  A[i] = (int*)malloc(sizeof(int) * N);
	//  B[i] = (int*)malloc(sizeof(int) * N);
	//  C[i] = (int*)malloc(sizeof(int) * N);
	//  //std::cout << i << std::endl;
	//}

	//for(int i = 0; i < N; ++i)
	//{
	//	for(int j = 0; j < N; ++j)
	//	{
	//		A[j * N + i] = i + j + 1;
	//		B[j * N + i] = i + j + 1;
	//	}
	//}

	for(int i = 0; i < N * N; ++i)
	{
		A[i] = i  + 1;
		B[i] = i  + 1;
		//std::cout << A[i] << " ";
		//if(i % 5 == 0)
		//	std::cout << std::endl;
	}


	// Add vectors in parallel.
    cudaStatus = addWithCuda1D2D(C, A, B, N * N);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    printf("A + B = {%d,%d,%d}\n",
        C[0], C[1], C[100]);
	std::cout << std::endl;

	// hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }


	// Add vectors in parallel.
    cudaStatus = addWithCuda2D2D(C, A, B, N * N);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    printf("A + B = {%d,%d,%d}\n",
        C[0], C[1], C[20]);



    return 0;
}
